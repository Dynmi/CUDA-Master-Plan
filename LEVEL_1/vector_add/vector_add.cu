#include "hip/hip_runtime.h"
/**
 * 
 * author@Haris Wang       
 * 	2020.10.7
 * 
 * */
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""



// initialize vector with random value
void init_vector(float *x, int n)
{
	for (int i=0; i<n; i++)
	{
		x[i] = (float)rand() % 1000;
	}
}


__global__ void vector_add(float *A, float *B, float *C)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	C[idx] = A[idx] + B[idx];
}


int main(void)
{	
		
	float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
	int n = 1024;

	// alloc host memory
	h_a = (float*)malloc(n * sizeof(float));
	h_b = (float*)malloc(n * sizeof(float));
	h_c = (float*)malloc(n * sizeof(float));

	// alloc gpu memory
	hipMalloc((void**)&d_a, n * sizeof(float));
	hipMalloc((void**)&d_b, n * sizeof(float));
	hipMalloc((void**)&d_c, n * sizeof(float));

	// copy data from host memory to gpu memory
	hipMemcpy(d_a, h_a, 4 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, 4 * sizeof(float), hipMemcpyHostToDevice);


	float time;
	hipEvent_t gpustart, gpustop;
	hipEventCreate(&gpustart);
	hipEventCreate(&gpustop);
	hipEventRecord(gpustart, 0);
	vector_add <<< ceil(n/512), 512 >>> (d_a, d_b, d_c);
	hipDeviceSynchronize();
	hipEventRecord(gpustop, 0);
	hipEventSynchronize(gpustop);
	hipEventElapsedTime(&time, gpustart, gpustop);
	hipEventDestroy(gpustart);
	hipEventDestroy(gpustop);

	// copy data from gpu memory to host memory
	hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

	printf("------------------------------------------------------------------------------------\n\n\n");
	printf("Time cost on GPU is %.2f ms \n",time);
	printf("------------------------------------------------------------------------------------\n\n\n");


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}